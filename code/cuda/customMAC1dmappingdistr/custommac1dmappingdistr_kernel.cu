#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <stdio.h>
#include <cstdint>
#include <chrono>

#define DEBUG_1D 0
#define DEBUG_THREAD_INFO_FLOAT32 0
#define DEBUG_THREAD_INFO_INT32 0
#define DEBUG_BITS 0
#define DEBUG_SEEDS 0

template <typename scalar_t>
__global__ void custommac1dmappingdistr_kernel(
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> input,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> weight,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> output,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> mapping_distr,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> mapping_distr_sorted_idx,
    int array_size,
    unsigned long long seed0
  )
{

  // handle access indices
  const int c = blockIdx.x * blockDim.x + threadIdx.x; // y
  const int d = blockIdx.y * blockDim.y + threadIdx.y; // x

  // make sure we don't modify memory regions outside of output
  if ((d < output.size(0)) && (c < output.size(1)))
  {
    // this is (c,d,e), we have as many threads as we have pixels in output out
    // each thread of out calculates a MAC (row of filter times column of input)

    // every thread is responsible for one sum, there are as many threads as mac sums in output
    output[d][c] = 0;
    float mult_result = 0;
    for(int i = 0; i < weight.size(1); i++)
    {
        //printf("Thread: (%d,%d,%d)\nWeight: %.4f, Input: %.4f\n", c, d, e, weight[c][i], input[d][i][e]);
        mult_result = weight[c][i] * input[d][i];

        output[d][c] += mult_result;
    }
  }
}

torch::Tensor custommac1dmappingdistr_cuda(
  torch::Tensor input,
  torch::Tensor weight,
  torch::Tensor output,
  torch::Tensor mapping_distr,
  torch::Tensor mapping_distr_sorted_idx,
  int array_size
) {
  // The number of thread blocks in a grid is usually dictated by the size of the data being processed, which typically exceeds the number of processors in the system.
  // dim3 threadsPerBlock(8,8,8)
  // <<<number of blocks per grid, number of threads ber block>>>
  // grid is created with enough blocks to have one thread per matrix element

  // https://devtalk.nvidia.com/default/topic/1028226/how-many-concurrent-threads-are-running-on-my-geforce-gtx-1080-ti-/
  const int output_size_x = output.size(1);
  const int output_size_y = output.size(0);
  int threads_x = 16; // per block, 16
  int threads_y = 16; // per block, 16

  #if DEBUG_1D
    threads_x = 1;
    threads_y = 1;
  #endif

  const dim3 threads(threads_x,threads_y);
  const dim3 blocks((output_size_x + threads_x - 1) / threads_x,
                    (output_size_y + threads_y - 1) / threads_y);

  // create a seed from the current time in nanoseconds
  auto now = std::chrono::system_clock::now();
  auto now_ms = std::chrono::time_point_cast<std::chrono::nanoseconds>(now);
  auto value = now_ms.time_since_epoch();
  unsigned long long seed0 = value.count();

  AT_DISPATCH_ALL_TYPES(input.type(), "custommac1dmappingdistr_cuda", ([&] {
    custommac1dmappingdistr_kernel<scalar_t><<<blocks, threads>>>(
        input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        weight.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        output.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        mapping_distr.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        mapping_distr_sorted_idx.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        array_size,
        seed0
    );
  }));

  return output;
}
